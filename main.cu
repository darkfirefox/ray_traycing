#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <time.h>
#include <float.h>

#include "helper.h"
#include "cuda_helper.h"
#include "hittable_list.h"
#include "sphere.h"

int main()
{
    int width = 1200;
    int height = 600;
    int tx = 16;
    int ty = 16;
    int ns = 100;
    dim3 blocsDim(width / tx + 1, height / ty + 1);
    dim3 threadsDim(tx, ty);
    hiprandState *d_state, *d_state_cw;

    std::cerr << "Rendering a " << width << "x" << height << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    size_t fb_size = width * height * sizeof(Vec3);

    Vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    checkCudaErrors(hipMalloc((void **)&d_state, width * height *sizeof(hiprandState)));
    checkCudaErrors(hipMalloc((void **)&d_state_cw, 1*sizeof(hiprandState)));

    curandInit<<<1,1>>>(d_state_cw);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    Hittable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, (22*22+1) * sizeof(Hittable *)));
    Hittable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(Hittable *)));
    Camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera *)));
    createWorld<<<1, 1>>>(d_list, d_world, d_camera, width, height, d_state_cw);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    renderInit<<<blocsDim, threadsDim>>>(width, height, d_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocsDim, threadsDim>>>(fb, width, height, ns, d_camera, d_world, d_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    writeImage(fb, height, width);

    checkCudaErrors(hipDeviceSynchronize());
    freeWorld<<<1, 1>>>(d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(fb));
    hipDeviceReset();
}